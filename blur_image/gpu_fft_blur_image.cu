#include "hip/hip_runtime.h"
#include "gpu_fft_blur_image.h"
#include "helper/cuda_check.h"
#include "helper/cufft_check.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core.hpp>
#include <cassert>

__global__ void complexPointwiseMulAndScale_kernel(hipfftComplex *a, hipfftComplex *b, int size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int batch = blockIdx.y;

    if (index < size)
    {
        float scale = 1.0f / (float)size;
        hipfftComplex c = hipCmulf(a[batch * size + index], b[batch * size + index]);
        b[batch * size + index] = make_hipFloatComplex(scale * hipCrealf(c), scale * hipCimagf(c));
    }
}

GpuBlurImage::GpuBlurImage(int image_rows, int image_cols, int max_images, int kernel_size)
    : NY(image_rows)
    , NX(image_cols)
    , max_images(max_images)
{
    if (kernel_size > std::min(NY, NX))
    {
        throw std::invalid_argument("Kernel size cannot be larger than the image dimensions");
    }
    
    int n[2] = {NY, NX};
    CUFFT_CHK(hipfftPlanMany(&plan, 2, n, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, max_images));

    CUDA_CHK(hipMalloc(&d_complex, sizeof(hipfftComplex) * NY * NX * max_images));
    CUDA_CHK(hipMallocManaged(&d_gaussian_kernel, sizeof(hipfftComplex) * NY * NX * max_images));

    std::vector<float> filter = createGaussianFilter(kernel_size);

    // Set d_gaussian_kernel[*].y to 0.f
    CUDA_CHK(hipMemset(d_gaussian_kernel, 0, sizeof(hipfftComplex) * NY * NX));

    // Set d_gaussian_kernel[*].x
    for( int i = 0; i < kernel_size; ++i )
    {
        CUDA_CHK(hipMemcpy2D(&d_gaussian_kernel[i * NX], sizeof(hipfftComplex), &filter[i * kernel_size], sizeof(float), sizeof(float), kernel_size, hipMemcpyHostToDevice));
    }

    hipfftHandle gaussian_plan;
    CUFFT_CHK(hipfftPlan2d(&gaussian_plan, NY, NX, HIPFFT_C2C));
    CUFFT_CHK(hipfftExecC2C(gaussian_plan, d_gaussian_kernel, d_gaussian_kernel, HIPFFT_FORWARD));
    CUDA_CHK(hipDeviceSynchronize());
    CUFFT_CHK(hipfftDestroy(gaussian_plan));

    // Copy the FFT gaussian kernel for every image in batch
    for (int i = 1; i < max_images; ++i)
    {
        CUDA_CHK(hipMemcpy(&d_gaussian_kernel[i * NY * NX], d_gaussian_kernel, sizeof(hipfftComplex) * NY * NX, hipMemcpyDeviceToDevice));
    }
}

GpuBlurImage::~GpuBlurImage()
{
    CUFFT_CHK(hipfftDestroy(plan));
    CUDA_CHK(hipFree(d_complex));
    CUDA_CHK(hipFree(d_gaussian_kernel));
}

void GpuBlurImage::blur(float *blurred_images, float *images, int num_images)
{
    assert(num_images <= max_images);
    const int total = NY * NX * num_images;

    // Set d_complex[*].y to 0.f
    CUDA_CHK(hipMemset(d_complex, 0, sizeof(hipfftComplex) * total));
    // Set d_complex[*].x
    CUDA_CHK(hipMemcpy2D(d_complex, sizeof(hipfftComplex), images, sizeof(float), sizeof(float), total, hipMemcpyHostToDevice));

    CUFFT_CHK(hipfftExecC2C(plan, d_complex, d_complex, HIPFFT_FORWARD));

    {
        dim3 block(256);
        dim3 grid((NY * NX + block.x + 1) / block.x, num_images);
        complexPointwiseMulAndScale_kernel<<<grid, block>>>(d_gaussian_kernel, d_complex, NY * NX);
    }

    CUFFT_CHK(hipfftExecC2C(plan, d_complex, d_complex, HIPFFT_BACKWARD));

    // Copy down real data
    CUDA_CHK(hipMemcpy2D(blurred_images, sizeof(float), d_complex, sizeof(hipfftComplex), sizeof(float), total, hipMemcpyDeviceToHost));
}

std::vector<float> GpuBlurImage::createGaussianFilter(int kernel_size)
{
    const int radius = kernel_size / 2;
    const float sigma = kernel_size / 4; // By default, radius of kernel = 2 * sigma
    const float q = 2.0 * sigma * sigma;
    float sum = 0.0; // sum is for normalization

    std::vector<float> kernel(kernel_size * kernel_size);

    for (int y = 0; y < kernel_size; y++)
    {
        for (int x = 0; x < kernel_size; x++)
        {
            float p = sqrt((x - radius) * (x - radius) + (y - radius) * (y - radius));
            kernel[y * kernel_size + x] = (exp(-(p * p) / q)) / (M_PI * q);
            sum += kernel[y * kernel_size + x];
        }
    }

    for (auto& v : kernel)
    {
        v /= sum; // normalize
    }

    printf("Gaussian filter:\n");
    for (int i = 0; i < kernel_size; ++i)
    {
        for (int j = 0; j < kernel_size; ++j)
        {
            printf("%f ", kernel[i * kernel_size + j]);
        }
        printf("\n");
    }

    return kernel;
}